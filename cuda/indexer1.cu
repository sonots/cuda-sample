
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NDIM 2

template <int8_t kNdim>
class IndexIterator {
public:
    __host__ __device__ void Set(int64_t i) {
        for (int8_t j = kNdim; --j >= 0;) {
            index_[j] = i % shape_[j];
            i /= shape_[j];
        }
    }

    __host__ __device__ int64_t* index() { return index_; }

private:
    //const int64_t* shape_;
    int64_t shape_[kNdim];
    //int64_t total_size_{};
    //int64_t raw_index_{};
    //int64_t step_{};
    int64_t index_[kNdim];
};

__global__ void test(IndexIterator<NDIM> i) {
    i.Set(1);
    printf("1\n", i.index()[0]);
}

int main() {
    IndexIterator<NDIM> i{};
    test<<<1,1>>>(i);
}
