// nvcc 036 sgemm .cu -lcublas

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define m 6 // a - mxk matrix
#define n 4 // b - kxn matrix
#define k 5 // c - mxn matrix

int main(void) {
    hipblasHandle_t handle; // CUBLAS context
    int i,j; // i-row valex, j-column valex
    float *a; // mxk matrix
    float *b; // kxn matrix
    float *c; // mxn matrix
    // unified memory for a,b,c
    hipMallocManaged(&a, m*k*sizeof(hipComplex));
    hipMallocManaged(&b, k*n*sizeof(hipComplex));
    hipMallocManaged(&c, m*n*sizeof(hipComplex));
    // define an mxk matrix a column by column
    int val=0; // a:
    for (i=0;i<m;i++){
        for (j=0;j<k;j++){
            a[IDX2C(i,j,m)] = (float)val++;
        }
    }
    printf ("a:\n");
    for (i=0;i<m;i++){
        for (j=0;j<k;j++){
            printf("%5.0f",a[IDX2C(i,j,m)]);
        }
        printf ("\n");
    }
    // define a kxn matrix b column by column
    val=0; // b:
    for (i=0;i<k;i++){
        for (j=0;j<n;j++){
            b[IDX2C(i,j,k)] = (float)val++;
        }
    }
    printf ("b:\n");
    for (i=0;i<k;i++){
        for (j=0;j<n;j++){
            printf("%5.0f",b[IDX2C(i,j,k)]);
        }
        printf ("\n");
    }
    // define an mxn matrix c column by column
    val=0; // c:
    for (i=0;i<m;i++){
        for (j=0;j<n;j++){
            c[IDX2C(i,j,m)] = (float)0;
        }
    }
    printf ("c:\n");
    for (i=0;i<m;i++){
        for (j=0;j<n;j++){
            printf("%5.0f",c[IDX2C(i,j,m)]);
        }
        printf ("\n");
    }
    hipblasCreate(&handle); // initialize CUBLAS context
    float al=1.0f; // al =1
    float bet=1.0f; // bet =1
    // matrix - matrix multiplication : c = al*a*b + bet *c
    // a -mxk matrix , b -kxn matrix , c -mxn matrix ;
    // al, bet - scalars
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,a,m,b,k,&bet,c,m);
    hipDeviceSynchronize();
    printf ("c after Sgemm :\n");
    for(i=0;i<m;i++){
        for(j=0;j<n;j++){
            printf("%7.0f",c[IDX2C(i,j,m)]); // print c after Sgemm
        }
        printf("\n");
    }
    hipFree(a); // free memory
    hipFree(b); // free memory
    hipFree(c); // free memory
    hipblasDestroy(handle); // destroy CUBLAS context
    return EXIT_SUCCESS ;
}
